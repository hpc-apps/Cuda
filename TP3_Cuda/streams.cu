/*
 * Streams.
 * Simple example to show how to overlap computation and communication.
 * In this case, the computation is negligible compared to the communication but
 * using the Nvidia visual profiler shows that communication from the host to the
 * device and from the device to the host may overlap.
 */

#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>


__global__ void vecadd( int * v0, int * v1, std::size_t size )
{
  auto tid = blockIdx.x * blockDim.x + threadIdx.x;
  if( tid < size )
  {
    v0[ tid ] += v1[ tid ];
  }
}


int main()
{
  std::size_t const size = 1000000;
  std::size_t const sizeb = size * sizeof( int );

  int * v0 = nullptr;
  int * v1 = nullptr;
	
  // Allocation on the host is done with the cudaMallocHost function.
  // It is mandatory for streams since the memory needs to be pinned
  // i.e. fixed in RAM and not swapable.
  hipHostMalloc( &v0, sizeb , hipHostMallocDefault);
  hipHostMalloc( &v1, sizeb, hipHostMallocDefault);
  
  for( std::size_t i = 0 ; i < size ; ++i )
  {
    v0[ i ] = v1[ i ] = i;
  }
  
  int * v0_d = nullptr;
  int * v1_d = nullptr;

  hipMalloc( &v0_d, sizeb );
  hipMalloc( &v1_d, sizeb );
  
  // Streams declaration.
  hipStream_t streams[ 2 ];

  // Creation.
  hipStreamCreate( &streams[ 0 ] );
  hipStreamCreate( &streams[ 1 ] );
  
  // Input vectors are send by halves.
  // The cudaMemcpyAsync function is used instead of the usual cudaMemcpy function
  // since it takes the stream as its last parameter.
  hipMemcpyAsync( v0_d, v0, size/2 * sizeof(int), hipMemcpyHostToDevice, streams[ 0 ] );
  hipMemcpyAsync( v1_d, v1, size/2 * sizeof(int), hipMemcpyHostToDevice, streams[ 0 ] );
  
  hipMemcpyAsync( v0_d+size/2, v0+size/2, size/2 * sizeof(int), hipMemcpyHostToDevice, streams[ 1 ] );
  hipMemcpyAsync( v1_d+size/2, v1+size/2, size/2 * sizeof(int), hipMemcpyHostToDevice, streams[ 1 ] );
  
  dim3 block( 1024 );
  dim3 grid( (size - 1) / block.x * block.x + 1 );
  
  // One kernel is launched in each stream.
  vecadd<<< 1, size/2, 0, streams[ 0 ] >>>( v0_d, v1_d, size/2 );

  vecadd<<< 1, size/2, 0, streams[ 1 ] >>>( v0_d+size/2, v1_d+size/2, size/2 );
 
  // Sending back the resulting vector by halves.
  hipMemcpyAsync( v0, v0_d, size/2 * sizeof(int), hipMemcpyDeviceToHost, streams[ 0 ] );
  hipMemcpyAsync( v0+size/2, v0_d+size/2, size/2 * sizeof(int), hipMemcpyDeviceToHost, streams[ 1 ] );
  
  // Synchronize everything.
  hipDeviceSynchronize();
  
  // Destroy streams.
  hipStreamDestroy( streams[ 0 ] );
  hipStreamDestroy( streams[ 1 ] );
     
  for( std::size_t i = 0 ; i < size ; ++i )
  {
    std::cout << v0[ i ] << std::endl;
  }
  
  hipFree( v0_d );
  hipFree( v1_d );
 
  hipHostFree( v0 );
  hipHostFree( v1 );
 
  return 0;
}
