
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>


__global__ void vecadd( int * v1, int * v2 )
{
  auto tid = threadIdx.x;

         v2[tid] += v1[tid];
	 
}


int main()
{
  std::vector< int > v1( 10 );
  std::vector< int > v2( 10 );
  hipError_t cudaStatus;
  hipError_t kernelStatus;

  int * v1_d = nullptr;
  int * v2_d = nullptr;

  for( std::size_t i = 0 ; i < v1.size() ; ++i )
  {
    v1[ i ] =  i;
    v2[ i ] =  i;
  }
  
  
  
  cudaStatus = hipMalloc( &v1_d, v1.size() * sizeof( int ) );
  if (cudaStatus != hipSuccess)
  {
	std::cout << "Error CudaMalloc v1_d"  << " ";
  }


  cudaStatus = hipMalloc( &v2_d, v2.size() * sizeof( int ) );
  if (cudaStatus != hipSuccess)
  {
	std::cout << "Error CudaMalloc v2_d" << " ";
  }


  cudaStatus= hipMemcpy(v1_d, v1.data(), v1.size() * sizeof( int ), hipMemcpyHostToDevice );
  if (cudaStatus  != hipSuccess)
  {
	  std::cout << "Error hipMemcpy v1_d - HostToDevice" << " ";
  }



  cudaStatus = hipMemcpy(v2_d, v2.data(), v2.size() * sizeof( int ), hipMemcpyHostToDevice );
  if (cudaStatus != hipSuccess)
   {
	   std::cout << "Error hipMemcpy v2_d - HotToDevice" << " ";
  }


  vecadd<<< 1, 10 >>>( v1_d, v2_d );
  kernelStatus = hipGetLastError();
   if ( kernelStatus != hipSuccess )
   {
	   std::cout << "CUDA Error"<< hipGetErrorString(kernelStatus) << " ";
  }


  cudaStatus = hipMemcpy(v2.data(),v2_d, v2.size() * sizeof( int ), hipMemcpyDeviceToHost );
  if (cudaStatus != hipSuccess) 
  {
	std::cout << "Error Cuda Memcpy v2_d DeviceToHost"  << " " ;
  }
 
 
  for (size_t idex = 0; idex < v1.size(); idex++)
    std::cout <<   v2[idex] << " ";
    std::cout << std::endl;



  hipFree( v1_d );
  hipFree( v2_d );

  return 0;
}
