
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>


__global__ void vecadd( int * v1, int * v2 )
{
  auto tid = threadIdx.x;

         v2[tid] += v1[tid];
	 
}


int main()
{
  int N=1024;
  std::vector< int > v1( N );
  std::vector< int > v2( N );
  hipError_t cudaStatus;
  hipError_t kernelStatus;
  float elapsedTime;
  hipEvent_t start, stop;

  int * v1_d = nullptr;
  int * v2_d = nullptr;



  hipEventCreate(&start);
  hipEventCreate(&stop);
  for( std::size_t i = 0 ; i < v1.size() ; ++i )
  {
    v1[ i ] =  i;
    v2[ i ] =  i;
  }
  
  
  
  cudaStatus = hipMalloc( &v1_d, v1.size() * sizeof( int ) );
  if (cudaStatus != hipSuccess)
  {
	std::cout << "Error CudaMalloc v1_d"  << " ";
  }


  cudaStatus = hipMalloc( &v2_d, v2.size() * sizeof( int ) );
  if (cudaStatus != hipSuccess)
  {
	std::cout << "Error CudaMalloc v2_d" << " ";
  }


  cudaStatus= hipMemcpy(v1_d, v1.data(), v1.size() * sizeof( int ), hipMemcpyHostToDevice );
  if (cudaStatus  != hipSuccess)
  {
	  std::cout << "Error hipMemcpy v1_d - HostToDevice" << " ";
  }



  cudaStatus = hipMemcpy(v2_d, v2.data(), v2.size() * sizeof( int ), hipMemcpyHostToDevice );
  if (cudaStatus != hipSuccess)
   {
	   std::cout << "Error hipMemcpy v2_d - HotToDevice" << " ";
  }

  hipEventRecord(start, 0);  
  vecadd<<< 1, N >>>( v1_d, v2_d );
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  std::cout << "Timing (ms) = " << elapsedTime << " ";
  std::cout << std::endl;



  kernelStatus = hipGetLastError();
   if ( kernelStatus != hipSuccess )
   {
	   std::cout << "CUDA Error"<< hipGetErrorString(kernelStatus) << " ";
  }


  cudaStatus = hipMemcpy(v2.data(),v2_d, v2.size() * sizeof( int ), hipMemcpyDeviceToHost );
  if (cudaStatus != hipSuccess) 
  {
	std::cout << "Error Cuda Memcpy v2_d DeviceToHost"  << " " ;
  }
 
 
//  for (size_t idex = 0; idex < v1.size(); idex++)
//    std::cout <<   v2[idex] << " ";
//    std::cout << std::endl;



  hipFree( v1_d );
  hipFree( v2_d );

  return 0;
}
