
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>


__global__ void vecadd( int * v1, int * v2 )
{
  auto tid = threadIdx.x;

         v2[tid] += v1[tid];
	 
}


int main()
{
  std::vector< int > v1( 10 );
  std::vector< int > v2( 10 );
  
  int * v1_d = nullptr;
  int * v2_d = nullptr;

  for( std::size_t i = 0 ; i < v1.size() ; ++i )
  {
    v1[ i ] =  i;
    v2[ i ] =  i;
  }
  
  hipMalloc( &v1_d, v1.size() * sizeof( int ) );
  hipMalloc( &v2_d, v2.size() * sizeof( int ) );

  hipMemcpy(v1_d, v1.data(), v1.size() * sizeof( int ), hipMemcpyHostToDevice );
  hipMemcpy(v2_d, v2.data(), v2.size() * sizeof( int ), hipMemcpyHostToDevice );
   

  vecadd<<< 1, 10 >>>( v1_d, v2_d );

  hipMemcpy(v2.data(),v2_d, v2.size() * sizeof( int ), hipMemcpyDeviceToHost );

 
 
  for (size_t idex = 0; idex < v1.size(); idex++)
    std::cout <<   v2[idex] << " ";
    std::cout << std::endl;



  hipFree( v1_d );
  hipFree( v2_d );

  return 0;
}
