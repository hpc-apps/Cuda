
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>


__global__ void vecswap( int * v1, int * v2  )
{
  auto tid = threadIdx.x;
  int  swap;
         swap = v1[tid];
	 v1[tid] = v2 [tid];
	 v2[tid] = swap ;
}


int main()
{
  std::vector< int > v1( 10 );
  std::vector< int > v2( 10 );
  
  int * v1_d = nullptr;
  int * v2_d = nullptr;

  for( std::size_t i = 0 ; i < v1.size() ; ++i )
  {
    v1[ i ] =  i;
    v2[ i ] =  v1.size()-i;
  }
  
  hipMalloc( &v1_d, v1.size() * sizeof( int ) );
  hipMalloc( &v2_d, v2.size() * sizeof( int ) );

  hipMemcpy(v1_d, v1.data(), v1.size() * sizeof( int ), hipMemcpyHostToDevice );
  hipMemcpy(v2_d, v2.data(), v2.size() * sizeof( int ), hipMemcpyHostToDevice );
   
  for (size_t idex = 0; idex < v1.size() ; idex++)
	std::cout << v1[idex] <<"/"<< v2[idex] << " ";
  	std::cout << std::endl;


  vecswap<<< 1, 10 >>>( v1_d, v2_d );

  hipMemcpy(v1.data(),v1_d, v1.size() * sizeof( int ), hipMemcpyDeviceToHost );
  hipMemcpy(v2.data(),v2_d, v2.size() * sizeof( int ), hipMemcpyDeviceToHost );

 
 
  for (size_t idex = 0; idex < v1.size(); idex++)
    std::cout << v1[idex] <<"//"<<  v2[idex] << " ";
    std::cout << std::endl;


  hipFree( v1_d );
  hipFree( v2_d );

  return 0;
}
