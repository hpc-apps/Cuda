/**
 * Detect the number of CUDA capable devices.
 */

#include <hip/hip_runtime.h>
#include <iostream>


int main()
{
  int count = 0;

  hipGetDeviceCount( &count );

  std::cout << count << " device(s) found.\n";
  
  return 0;
}
