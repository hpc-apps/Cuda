#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>

__global__ void grayscale( unsigned char * rgb, unsigned char * g, std::size_t cols, std::size_t rows ) {
  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  auto j = blockIdx.y * blockDim.y + threadIdx.y;
  if( i < cols && j < rows ) {
    g[ j * cols + i ] = (
			 307 * rgb[ 3 * ( j * cols + i ) ]
			 + 604 * rgb[ 3 * ( j * cols + i ) + 1 ]
			 + 113 * rgb[  3 * ( j * cols + i ) + 2 ]
			 ) / 1024;
  }
}

int main()
{
  cv::Mat m_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED );
  auto rgb = m_in.data;
  auto rows = m_in.rows;
  auto cols = m_in.cols;
  std::vector< unsigned char > g( rows * cols );
  cv::Mat m_out( rows, cols, CV_8UC1, g.data() );
  unsigned char * rgb_d;
  unsigned char * g_d;
  hipMalloc( &rgb_d, 3 * rows * cols );
  hipMalloc( &g_d, rows * cols );
  hipMemcpy( rgb_d, rgb, 3 * rows * cols, hipMemcpyHostToDevice );
  dim3 t( 32, 32 );
  dim3 b( ( cols - 1) / t.x + 1 , ( rows - 1 ) / t.y + 1 );
  grayscale<<< b, t >>>( rgb_d, g_d, cols, rows );
  hipMemcpy( g.data(), g_d, rows * cols, hipMemcpyDeviceToHost );
  cv::imwrite( "out.jpg", m_out );
  hipFree( rgb_d);
  hipFree( g_d);
  return 0;
}
